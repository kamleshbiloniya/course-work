
#include <hip/hip_runtime.h>
#include<stdio.h>
#include<stdlib.h>
#include<sys/time.h>




#define CUDA_ERROR_EXIT(str) do{\
                                    hipError_t err = hipGetLastError();\
                                    if( err != hipSuccess){\
                                             printf("Cuda Error: '%s' for %s\n", hipGetErrorString(err), str);\
                                             exit(-1);\
                                    }\
                             }while(0);
#define TDIFF(start, end) ((end.tv_sec - start.tv_sec) * 1000000UL + (end.tv_usec - start.tv_usec))
__global__ void XOR(int *in,int *out){
  extern __shared__ int sm[];             //shared memory
	
  unsigned int tid=threadIdx.x;
  unsigned int i=blockIdx.x*blockDim.x+threadIdx.x;
	//printf("i=%d\n",i);
  sm[tid]=in[i];
  __syncthreads();                              //snchronizes all threads in a block
  for(unsigned int j=1;j<blockDim.x;j*=2){    //ech thread loads one element from global to shared memory
    if(tid%(2*j)==0){
      sm[tid]=sm[tid]^sm[tid+j];
    }
  }
    __syncthreads();
  
	//printf("till now ??\n");
  if(tid==0){//printf("debuger\n");
	out[blockIdx.x]=sm[0];
	//printf("from block: %d\n",out[0]);
	}
}


int main(int argc, char **argv){
  struct timeval start,end,t_start,t_end;
  unsigned int num_ele,seed;
  int *harr,*darr;
   int result=0;
  int *hout,*dout;
  int th_p_block=10;
  int b_p_grid;



  num_ele=atoi(argv[1]);
  seed=atoi(argv[2]);
  b_p_grid=(num_ele+th_p_block-1)/th_p_block;

  harr=(int*)malloc(num_ele*sizeof(int));
   if(!harr){
        perror("malloc");
        exit(-1);
   }



  hout=(int*)malloc(b_p_grid*sizeof(int));
 

  srand(seed);
  for(int i=0;i<num_ele;i++){
   //harr[i]=i+1;
   harr[i]=random()%100;
  }

  gettimeofday(&t_start,NULL);

  hipMalloc(&darr,num_ele*sizeof(int));
  CUDA_ERROR_EXIT("hipMalloc");

  hipMalloc(&dout,b_p_grid*sizeof(int));
  CUDA_ERROR_EXIT("hipMalloc");

	//printf("ok\n");
  hipMemcpy(darr,harr,num_ele*sizeof(int),hipMemcpyHostToDevice);
  CUDA_ERROR_EXIT("hipMemcpy");
    //printf("after memcpy\n");
  //int th_p_block=10;
  //int b_p_grid=(num_ele+th_p_block-1)/th_p_block;
  gettimeofday(&start,NULL);
  XOR<<<b_p_grid, th_p_block,2048>>>(darr,dout);
  CUDA_ERROR_EXIT("kernel invocation");
	gettimeofday(&end,NULL);
    //printf("before memcpy2 D to H\n");
  hipMemcpy(hout,dout,b_p_grid*sizeof(int),hipMemcpyDeviceToHost);
   CUDA_ERROR_EXIT("memcpy");
	//printf("done\n");
  
for(int i=0;i<b_p_grid;i++){
     result=result^hout[i];
	//printf("res=%d harr=%d\n",result,harr[i]);
}
   printf("xor=%d\n", result);
   gettimeofday(&t_end,NULL);
printf("Total time = %ld microsecs Processsing =%ld microsecs\n", TDIFF(t_start, t_end), TDIFF(start, end));

   free(harr);
   free(hout);
   hipFree(darr);
   hipFree(dout);

  return 0;
}
